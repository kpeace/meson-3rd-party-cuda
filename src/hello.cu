#include <iostream>
#include <hip/hip_runtime.h>


__global__ 
void hello(int *b) 
{
    *b = 43;
}


int main(){


    int *pd;
    hipMalloc(&pd, sizeof(int));
    //pd = (int*)malloc(sizeof(int));
    hello<<<1, 1>>>(pd);
    hipDeviceSynchronize();

    int res = 0;
    hipMemcpy(&res, pd, sizeof(int), hipMemcpyDeviceToHost );
    hipFree(pd);
    //res = *pd;

    std::cout << "hello world: " << res << std::endl;
    return 0;
}
