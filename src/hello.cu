#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <fmt/format.h>


__global__ 
void hello(int *b) 
{
    *b = 43;
}


int main(){


    int *pd;
    hipMalloc(&pd, sizeof(int));
    //pd = (int*)malloc(sizeof(int));
    hello<<<1, 1>>>(pd);
    hipDeviceSynchronize();

    int res = 0;
    hipMemcpy(&res, pd, sizeof(int), hipMemcpyDeviceToHost );
    hipFree(pd);
    //res = *pd;

    std::cout << fmt::format("hello {} world: ", "formated") << res << std::endl;
    return 0;
}
